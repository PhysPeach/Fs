//positionFile format
//diam1 diam2...
//
//t K U p1_x p1_y p1_vx p1_vy p2_x...
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>

#define NT 1024
#define NB 512

typedef unsigned int uint;

template <typename T>
T pow_tmp(T x, int y){
	T xx = (T)1;
	if (y > 0){
		for (int i = 1; i <= y; i++) {
			xx *= x;
		}
	}
	else{
		for (int i = -1; i >= y; i--) {
			xx /= x;
		}
	}

	return xx;
}

//---settings---//
const uint D = 2;
uint ID;
uint NP;
double T;
double tmax;
//--------------//

const double pi = 4 * atan(1.0);
__global__ void coskr(double *coskr, double *x, double k, uint l, uint d){
    uint i_block = blockIdx.x;
    uint i_local = threadIdx.x;
    uint i_global = i_block * blockDim.x + i_local;
    for(uint i = i_global; i < l; i += NB*NT){
        coskr[i] = cos(k*(x[i]-x[i%d]));
    }
}
__global__ void reductionFs(double *out, double *coskr, uint l){
    uint i_block = blockIdx.x;
    uint i_local = threadIdx.x;
    uint i_global = i_block * blockDim.x + i_local;

    __shared__ double f[NT];

    uint remain, reduce;
    uint ib = i_block;
    for(uint i = i_global; i < l; i += NB*NT){
        f[i_local] = coskr[i];
        __syncthreads();

        for(uint j = NT; j > 1; j = remain){
            reduce = j >> 1;
            remain = j - reduce;
            if((i_local < reduce) && (i + remain < l)){
                f[i_local] += f[i_local+remain];
            }
            __syncthreads();
        }
        if(i_local == 0){
            out[ib] = f[0];
        }
        __syncthreads();
        ib += NB;
    }
}

int main(int argc, char** argv){
    ID = atoi(argv[1]);
    NP = atoi(argv[2]);
	T = atof(argv[3]);
    uint timescale= atoi(argv[4]);
    tmax = pow_tmp(2., timescale);
    
    std::cout << "---settings---" << std::endl;
    std::cout << "ID: [1, " << ID << "]" << std::endl;
    std::cout << "D: " << D << std::endl;
    std::cout << "NP: " << NP << std::endl;
    std::cout << "T: " << T << std::endl;
    std::cout << "timescale: " << timescale << std::endl;
    std::cout << "--------------" << std::endl << std::endl;

    //Variables
    double *diam;
    double *x, *x_dev;
    double *coskr_dev[2];
    double *t, *dt;
    diam = new double[NP];

    const double a0 = 1.;
    const double a1 = a0 * 1;
    const double a2 = a0 * 1.4;
    double dnsty = 0.8;
    double L = sqrt((double)NP/dnsty);

    uint Nt;

    //find dt, diam
    std::ostringstream positionName0;
	positionName0 << "../../pos/N"<< argv[2] << "/T" << argv[3] << "/posBD_N" << argv[2] << "_T" << argv[3] << "_id1.data";
	std::ifstream positionFile;
	positionFile.open(positionName0.str().c_str());
	std::cout << "Loading " << positionName0.str() << " for find dt, diam" << std::endl;
	double t1, t2, DA, DK, DU, DX, DV;
	for (uint n = 0; n < NP; n++) {
		positionFile >> diam[n];
	}
	positionFile >> t1 >> DK >> DU;
	for (int n = 0; n < NP; n++) {
        positionFile >> DX >> DX >> DV >> DV;
	}
	positionFile >> t2;
    positionFile.close();
    std::cout << "dt = " << t2-t1 << std::endl;
    Nt = 0;
	double ttmp = 10 * (t2-t1);
	while (ttmp < tmax) {
		ttmp *= 1.1;
		Nt++;
    }

    //newMemory
    x = new double[ID*Nt*NP*D];
    t = new double[Nt];
	dt = new double[Nt - 1];
    hipMalloc((void**)&x_dev, ID * Nt * NP * D * sizeof(double));
    hipMalloc((void**)&coskr_dev[0], ID * Nt * NP * D * sizeof(double));
    hipMalloc((void**)&coskr_dev[1], ID * Nt * NP * D * sizeof(double));

    //loadFile
    for (short i = 0; i < ID; i++){
        // positionFile: t pi_x pi_y...
        std::ostringstream positionName;
        positionName << "../../pos/N"<< argv[2] << "/T" << argv[3];
        positionName << "/posBD_N" << argv[2] << "_T" << argv[3] << "_id" << i+1 << ".data";
	    positionFile.open(positionName.str().c_str());
		std::cout << "Loading " << positionName.str() << "..." << std::endl;
    	for (int n = 0; n < NP; n++) {
	    	positionFile >> DA;
    	}
		for (int nt = 0; nt < Nt; nt++){
    			positionFile >> t[nt] >> DK >> DU;
			for (int n = 0; n < NP; n++){
                positionFile >> x[nt*ID*NP*D + i*NP*D + n*D];
                positionFile >> x[nt*ID*NP*D + i*NP*D + n*D + 1];
                positionFile >> DV >> DV;
			}
		}
		positionFile.close();
		std::cout << " -> done" << std::endl;
    }
    hipMemcpy(x_dev, x, Nt*ID*NP*D * sizeof(double), hipMemcpyHostToDevice);

	for (short nt = 1; nt <= Nt - 1; nt++){
		dt[nt - 1] = t[nt] - t[0];
    }
    
    //analise
    double k = 2 * pi / a1;
    int q1 = (int)(k*L / (2 * pi));
	double k1 = 2 * pi*q1 / L;
	double k2 = 2 * pi*(q1 + 1) / L;
    if ((k - k1) <= (k2 - k))
        k = k1;
    else
        k = k2;
    
    std::cout << "Recording Fs_N" << argv[2] << "_T_" << argv[3] << "_k2pi..." << std::endl;
    std::ostringstream fsName;
	fsName << "./data/Fs_N" << argv[2] << "_T" << argv[3] << "_k2pi.data";
	std::ofstream fsFile;
    fsFile.open(fsName.str().c_str());
    double *Fs;
    Fs = new double[Nt-1];

    coskr<<<NB,NT>>>(coskr_dev[0], x_dev, k, Nt*ID*NP*D, ID*NP*D);
    uint flip;
    for(uint nt = 1; nt <= Nt - 1; nt++){
        flip = 0;
        for(uint l = ID*NP*D; l > 1; l = (l+NT-1)/NT){
            flip = !flip;
            reductionFs<<<NB,NT>>>(&coskr_dev[flip][nt*ID*NP*D], &coskr_dev[!flip][nt*ID*NP*D], l);
        }
        hipMemcpy(&Fs[nt - 1], &coskr_dev[flip][nt*ID*NP*D], sizeof(double), hipMemcpyDeviceToHost);
    }
    for (int nt = 1; nt <= Nt - 1; nt++){
		fsFile << dt[nt - 1] << " " << Fs[nt - 1]/(ID*NP*D) << std::endl;
    }
    delete[] Fs;
    fsFile.close();
    
    
    //deleteMemory
    delete[] diam;
    delete[] x;
    delete[] t;
    delete[] dt;
    hipFree(x_dev);
    hipFree(coskr_dev[0]);
    hipFree(coskr_dev[1]);
    return 0;
}